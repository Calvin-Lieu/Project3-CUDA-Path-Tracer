#include "hip/hip_runtime.h"
#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <climits>
#include <utility>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <thrust/sequence.h>
#include <thrust/gather.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/device_ptr.h>
#include <thrust/binary_search.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"
#include "pathHelpers.h"
#include "sortKeys.h"
#include "directLighting.h"
#include "bvh.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err)
    {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file)
    {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
    getchar();
#endif
    exit(EXIT_FAILURE);
#endif
}

// Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;

// Sorting/reorder buffers
static uint32_t* dev_matKeys = nullptr;
static int* dev_indices = nullptr;
static PathSegment* dev_paths_alt = nullptr;
static ShadeableIntersection* dev_intersections_alt = nullptr;

static int* dev_lightGeomIdx = nullptr;
static int hst_numLights = 0;

static BVH dev_bvh;
static bool bvhBuilt = false;

static TriangleMeshData* dev_meshes = nullptr;
static int numMeshes = 0;

void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

void pathtraceInit(Scene* scene)
{
    hst_scene = scene;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // For sort
    hipMalloc(&dev_matKeys, pixelcount * sizeof(uint32_t));
    hipMalloc(&dev_indices, pixelcount * sizeof(int));
    hipMalloc(&dev_paths_alt, pixelcount * sizeof(PathSegment));
    hipMalloc(&dev_intersections_alt, pixelcount * sizeof(ShadeableIntersection));

    if (!scene->meshes.empty()) {
        numMeshes = scene->meshes.size();
        hipMalloc(&dev_meshes, numMeshes * sizeof(TriangleMeshData));

        std::vector<TriangleMeshData> hostMeshes(numMeshes);

        for (int i = 0; i < numMeshes; ++i) {
            const auto& hostMesh = scene->meshes[i];

            // Allocate and copy vertex data
            hipMalloc(&hostMeshes[i].vertices, hostMesh.vertices.size() * sizeof(float));
            hipMemcpy(hostMeshes[i].vertices, hostMesh.vertices.data(),
                hostMesh.vertices.size() * sizeof(float), hipMemcpyHostToDevice);

            // Allocate and copy normal data
            hipMalloc(&hostMeshes[i].normals, hostMesh.normals.size() * sizeof(float));
            hipMemcpy(hostMeshes[i].normals, hostMesh.normals.data(),
                hostMesh.normals.size() * sizeof(float), hipMemcpyHostToDevice);

            // Allocate and copy index data
            hipMalloc(&hostMeshes[i].indices, hostMesh.indices.size() * sizeof(unsigned int));
            hipMemcpy(hostMeshes[i].indices, hostMesh.indices.data(),
                hostMesh.indices.size() * sizeof(unsigned int), hipMemcpyHostToDevice);
            hostMeshes[i].triangleCount = hostMesh.indices.size() / 3;
        }

        hipMemcpy(dev_meshes, hostMeshes.data(), numMeshes * sizeof(TriangleMeshData), hipMemcpyHostToDevice);
    }
    else {
        // No meshes to load
        dev_meshes = nullptr;
        numMeshes = 0;
    }

    // Build emissive geoms list and copy to device
    
    std::vector<int> lightIdx;
    lightIdx.reserve(scene->geoms.size());
    for (int i = 0; i < (int)scene->geoms.size(); ++i) {
        int mid = scene->geoms[i].materialid;
        if (scene->materials[mid].emittance > 0.0f)
            lightIdx.push_back(i);
    }
    hst_numLights = (int)lightIdx.size();
    if (hst_numLights > 0) {
        hipMalloc(&dev_lightGeomIdx, hst_numLights * sizeof(int));
        hipMemcpy(dev_lightGeomIdx, lightIdx.data(),
            hst_numLights * sizeof(int), hipMemcpyHostToDevice);
    }
    //printf("Found %d lights\n", hst_numLights);
    //for (int i = 0; i < hst_numLights; i++) {
    //    printf("  Light %d: geom index %d\n", i, lightIdx[i]);
    //}

    // Build BVH
    dev_bvh = BVHBuilder::build(scene->geoms, scene->meshes);
    bvhBuilt = true;

    checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{
    hipFree(dev_image);
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);

    hipFree(dev_matKeys);
    hipFree(dev_indices);
    hipFree(dev_paths_alt);
    hipFree(dev_intersections_alt);
    hipFree(dev_lightGeomIdx);

    if (dev_meshes) {
        TriangleMeshData* hostMeshes = new TriangleMeshData[numMeshes];
        hipMemcpy(hostMeshes, dev_meshes, numMeshes * sizeof(TriangleMeshData), hipMemcpyDeviceToHost);

        for (int i = 0; i < numMeshes; ++i) {
            if (hostMeshes[i].vertices) hipFree(hostMeshes[i].vertices);
            if (hostMeshes[i].normals) hipFree(hostMeshes[i].normals);
            if (hostMeshes[i].indices) hipFree(hostMeshes[i].indices);
        }

        delete[] hostMeshes;
        hipFree(dev_meshes);
        dev_meshes = nullptr;
        numMeshes = 0;
    }

    BVHBuilder::free(dev_bvh);
    checkCUDAError("pathtraceFree");
}

// Generate PathSegments with rays from the camera into the scene
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment& segment = pathSegments[index];

        segment.ray.origin = cam.position;
        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);
        segment.prevBsdfPdf = 0.0f;
        segment.prevWasDelta = 0;

        // 4x4 stratified jitter per iteration
        const int S = 4;
        unsigned s = (iter - 1) % (S * S);
        int sx = s % S, sy = s / S;

        thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
        thrust::uniform_real_distribution<float> u01(0, 1);
        float jx = (sx + u01(rng)) / S - 0.5f;
        float jy = (sy + u01(rng)) / S - 0.5f;

        segment.ray.direction = glm::normalize(
            cam.view
            - cam.right * cam.pixelLength.x * ((x + jx) - cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * ((y + jy) - cam.resolution.y * 0.5f));

        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
    }
}

// Slightly optimized intersection kernel
__global__ void computeIntersections(
    int depth,
    int num_paths,
    const PathSegment* __restrict__ pathSegments,
    const Geom* __restrict__ geoms,
    int geoms_size,
    const TriangleMeshData* __restrict__ meshes,
    ShadeableIntersection* __restrict__ intersections)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths) return;

    const Ray ray = pathSegments[idx].ray;

    float t_min = 1e20f;
    int   hit_i = -1;
    glm::vec3 n_best = glm::vec3(0.0f);

    glm::vec3 I_tmp, N_tmp;
    bool outside;

#pragma unroll 1
    for (int i = 0; i < geoms_size; ++i)
    {
        const Geom& g = geoms[i];
        float t = -1.0f;

        if (g.type == CUBE) {
            t = boxIntersectionTest(g, ray, I_tmp, N_tmp, outside);
        }
        else if (g.type == SPHERE) {
            t = sphereIntersectionTest(g, ray, I_tmp, N_tmp, outside);
        }
        else if (g.type == TRIANGLE_MESH && g.meshIndex >= 0) {
            t = meshIntersectionTest(g, meshes[g.meshIndex], ray, I_tmp, N_tmp, outside);
        }

        if (t > 0.0f && t < t_min) {
            t_min = t;
            hit_i = i;
            n_best = N_tmp;
        }
    }

    ShadeableIntersection out;
    out.t = (hit_i < 0) ? -1.0f : t_min;
    if (hit_i >= 0) {
        out.materialId = geoms[hit_i].materialid;
        out.surfaceNormal = n_best;
        out.geomId = hit_i;
    }
    intersections[idx] = out;
}

__global__ void computeIntersectionsBVH(
    int depth, int num_paths,
    const PathSegment* __restrict__ pathSegments,
    const Geom* __restrict__ geoms,
    const BVHNode* __restrict__ bvhNodes,
    const BVHPrimitive* __restrict__ primitives,
    const TriangleMeshData* __restrict__ meshes,
    ShadeableIntersection* __restrict__ intersections)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths) return;

    const Ray ray = pathSegments[idx].ray;

    float t_min = 1e20f;
    int hit_i = -1;
    glm::vec3 n_best = glm::vec3(0.0f);

    glm::vec3 I_tmp, N_tmp;
    bool outside;

    // Stack for iterative BVH traversal
    int stack[64];
    int stackPtr = 0;
    stack[stackPtr++] = 0; // Start with root

    while (stackPtr > 0) {
        int nodeIdx = stack[--stackPtr];
        const BVHNode& node = bvhNodes[nodeIdx];

        // Test AABB
        if (!intersectAABB(ray, node.aabbMin, node.aabbMax))
            continue;

        if (node.leftChild == -1) {
            // Leaf node - test primitives
            for (int i = 0; i < node.primCount; i++) {
                const BVHPrimitive& prim = primitives[node.primStart + i];
                const Geom& g = geoms[prim.geomIndex];

                float t = -1.0f;

                if (prim.type == PRIM_GEOM) {
                    // Test whole geometry (sphere or cube)
                    if (g.type == CUBE) {
                        t = boxIntersectionTest(g, ray, I_tmp, N_tmp, outside);
                    }
                    else if (g.type == SPHERE) {
                        t = sphereIntersectionTest(g, ray, I_tmp, N_tmp, outside);
                    }
                }
                else {
                    // Test single triangle
                    t = singleTriangleIntersectionTest(g, meshes[g.meshIndex],
                        prim.triangleIndex, ray,
                        I_tmp, N_tmp, outside);
                }

                if (t > 0.0f && t < t_min) {
                    t_min = t;
                    hit_i = prim.geomIndex;
                    n_best = N_tmp;
                }
            }
        }
        else {
            // Interior node - add children to stack
            if (stackPtr < 62) {  // Leave room for both children
                stack[stackPtr++] = node.leftChild;
                stack[stackPtr++] = node.rightChild;
            }
        }
    }

    ShadeableIntersection out;
    out.t = (hit_i < 0) ? -1.0f : t_min;
    if (hit_i >= 0) {
        out.materialId = geoms[hit_i].materialid;
        out.surfaceNormal = n_best;
        out.geomId = hit_i;
    }
    intersections[idx] = out;
}

// Per-class shading over contiguous spans
__global__ void shadeEmissiveRange(
    int n, int depth,
    const ShadeableIntersection* __restrict__ isects,
    const Material* __restrict__ materials,
    PathSegment* paths,
    const Geom* __restrict__ geoms,
    const int* __restrict__ lightIdx,
    int numLights,
    glm::vec3* __restrict__ image)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    const ShadeableIntersection& isect = isects[i];
    PathSegment& path = paths[i];
    const Material& m = materials[isect.materialId];

    glm::vec3 Le = m.color * m.emittance;
    glm::vec3 contrib = evalEmissiveWithMIS(path, isect, Le, depth, geoms, lightIdx, numLights);

    atomicAddVec3(image, path.pixelIndex, contrib);

    // Terminate path
    path.color = glm::vec3(0.0f);
    path.remainingBounces = 0;
}
__global__ void shadeDiffuseRange(
    int iter, int n, int depth, int useRR, int useNEE,
    const ShadeableIntersection* __restrict__ isects,
    const Material* __restrict__ materials,
    PathSegment* paths,
    const Geom* __restrict__ geoms, int ngeoms,
    const int* __restrict__ lightIdx, int numLights,
    glm::vec3* __restrict__ image)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    const ShadeableIntersection isect = isects[i];
    PathSegment& ps = paths[i];
    if (ps.remainingBounces <= 0 || isect.t < 0.f) return;

    thrust::default_random_engine rng =
        makeSeededRandomEngine(iter, paths[i].pixelIndex, depth);

    const glm::vec3 P = ps.ray.origin + ps.ray.direction * isect.t;
    const glm::vec3 N = isect.surfaceNormal;
    const glm::vec3 wo = -ps.ray.direction;
    const Material& m = materials[isect.materialId];

    // NEE only for diffuse surfaces
    if (useNEE && numLights > 0 && isDiffuse(m)) {
        const glm::vec3 albedoTimesThroughput = m.color * ps.color;
        addDirectLighting_NEEDiffuse(
            P, N, wo,
            materials,
            geoms, ngeoms,
            lightIdx, numLights,
            albedoTimesThroughput,
            ps.pixelIndex,
            image,
            rng);
    }


    scatterRay(ps, P, N, m, rng);

    if (useRR) applyRussianRoulette(ps, depth, 3, 0.05f, rng);
}

__global__ void gatherTerminated(int n, glm::vec3* image, PathSegment* paths)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    if (paths[i].remainingBounces <= 0) {
        image[paths[i].pixelIndex] += paths[i].color;
        paths[i].color = glm::vec3(0.0f);
    }
}

__global__ void shadeMaterials(
    int iter, int num_paths, int depth, int useRR, int useNEE,
    ShadeableIntersection* __restrict__ isects,
    PathSegment* paths,
    Material* __restrict__ materials,
    const Geom* __restrict__ geoms, int ngeoms,
    const int* __restrict__ lightIdx, int numLights,
    glm::vec3* __restrict__ image)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths) return;

    const ShadeableIntersection isect = isects[idx];
    PathSegment& ps = paths[idx];

    if (ps.remainingBounces <= 0) { ps.color = glm::vec3(0); return; }
    if (isect.t < 0.0f) { ps.color = glm::vec3(0); ps.remainingBounces = 0; return; }

    Material& m = materials[isect.materialId];

    if (m.emittance > 0.0f) {
        glm::vec3 Le = m.color * m.emittance;
        glm::vec3 contrib = evalEmissiveWithMIS(ps, isect, Le, depth, geoms, lightIdx, numLights);
        atomicAddVec3(image, ps.pixelIndex, contrib);
        ps.color = glm::vec3(0);
        ps.remainingBounces = 0;
        return;
    }

    thrust::default_random_engine rng =
        makeSeededRandomEngine(iter, paths[idx].pixelIndex, depth);

    const glm::vec3 P = ps.ray.origin + ps.ray.direction * isect.t;
    const glm::vec3 N = isect.surfaceNormal;
    const glm::vec3 wo = -ps.ray.direction;

    if (useNEE && numLights > 0 && isDiffuse(m)) {
        const glm::vec3 albedoTimesThroughput = m.color * ps.color;
        addDirectLighting_NEEDiffuse(
            P, N, wo,
            materials,
            geoms, ngeoms,
            lightIdx, numLights,
            albedoTimesThroughput,
            ps.pixelIndex,
            image,
            rng);
    }

    scatterRay(ps, P, N, m, rng);

    if (useRR) applyRussianRoulette(ps, depth, 3, 0.05f, rng);
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] += iterationPath.color;
    }
}

void pathtrace(uchar4* pbo, int frame, int iter)
{
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    const int blockSize1d = 128;

    generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");

    int depth = 0;
    int num_paths = cam.resolution.x * cam.resolution.y;

    bool iterationComplete = false;
    while (!iterationComplete)
    {
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        const int useBVH = (guiData && guiData->UseBVH) ? 1 : 0;
        if (useBVH && bvhBuilt) {
            computeIntersectionsBVH << <numblocksPathSegmentTracing, blockSize1d >> > (
                depth, num_paths, dev_paths, dev_geoms,
                dev_bvh.nodes, dev_bvh.primitives, dev_meshes, dev_intersections);
        }
        else {
            computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
                depth, num_paths, dev_paths, dev_geoms,
                hst_scene->geoms.size(), dev_meshes, dev_intersections);
        }
        checkCUDAError("trace one bounce");
        hipDeviceSynchronize();

        {
            auto zip_begin = thrust::make_zip_iterator(thrust::make_tuple(dev_paths, dev_intersections));
            auto zip_end = zip_begin + num_paths;

            thrust::for_each(thrust::device, zip_begin, zip_end, MarkMissDead{});
            auto zip_new_end = thrust::remove_if(thrust::device, zip_begin, zip_end, IsDeadTuple{});
            num_paths = static_cast<int>(zip_new_end - zip_begin);
        }

        depth++;

        if (num_paths == 0) {
            iterationComplete = true;
            if (guiData) guiData->TracedDepth = depth;
            break;
        }

        const int useRR = (guiData && guiData->UseRussianRoulette) ? 1 : 0;
        const int useNEE = (guiData && guiData->UseDirectLighting) ? 1 : 0;

        bool doSort = !guiData ? true : guiData->SortByMaterial;
        if (doSort) {
            // --- simple: sort by material id only ---
            buildMaterialKeys << <numblocksPathSegmentTracing, blockSize1d >> > (
                num_paths, dev_intersections, dev_matKeys);
            checkCUDAError("build material keys");

            thrust::sequence(thrust::device, dev_indices, dev_indices + num_paths);

            thrust::sort_by_key(thrust::device,
                dev_matKeys, dev_matKeys + num_paths,
                dev_indices);

            thrust::gather(thrust::device, dev_indices, dev_indices + num_paths,
                dev_paths, dev_paths_alt);
            thrust::gather(thrust::device, dev_indices, dev_indices + num_paths,
                dev_intersections, dev_intersections_alt);

            std::swap(dev_paths, dev_paths_alt);
            std::swap(dev_intersections, dev_intersections_alt);
            checkCUDAError("reorder by material id");
        }
        
        shadeMaterials << <numblocksPathSegmentTracing, blockSize1d >> > (
            iter, num_paths, depth, useRR, useNEE,
            dev_intersections, dev_paths, dev_materials,
            dev_geoms, (int)hst_scene->geoms.size(),
            dev_lightGeomIdx, hst_numLights,
            dev_image);
        checkCUDAError("mega shading");
        

        gatherTerminated << <numblocksPathSegmentTracing, blockSize1d >> > (
            num_paths, dev_image, dev_paths);

        auto newEnd = thrust::remove_if(
            thrust::device, dev_paths, dev_paths + num_paths, IsDeadPath{});

        num_paths = static_cast<int>(newEnd - dev_paths);
        iterationComplete = (num_paths == 0) || (depth >= traceDepth);

        if (guiData != NULL)
        {
            guiData->TracedDepth = depth;
        }
    }
      
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather << <numBlocksPixels, blockSize1d >> > (num_paths, dev_image, dev_paths);

    sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);

    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
