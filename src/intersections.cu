#include "hip/hip_runtime.h"
#include "intersections.h"
#define GLM_ENABLE_EXPERIMENTAL
#include <glm/gtx/intersect.hpp>

__host__ __device__ float boxIntersectionTest(
    Geom box,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    Ray q;
    q.origin    =                multiplyMV(box.inverseTransform, glm::vec4(r.origin   , 1.0f));
    q.direction = glm::normalize(multiplyMV(box.inverseTransform, glm::vec4(r.direction, 0.0f)));

    float tmin = -1e38f;
    float tmax = 1e38f;
    glm::vec3 tmin_n;
    glm::vec3 tmax_n;
    for (int xyz = 0; xyz < 3; ++xyz)
    {
        float qdxyz = q.direction[xyz];
        /*if (glm::abs(qdxyz) > 0.00001f)*/
        {
            float t1 = (-0.5f - q.origin[xyz]) / qdxyz;
            float t2 = (+0.5f - q.origin[xyz]) / qdxyz;
            float ta = glm::min(t1, t2);
            float tb = glm::max(t1, t2);
            glm::vec3 n;
            n[xyz] = t2 < t1 ? +1 : -1;
            if (ta > 0 && ta > tmin)
            {
                tmin = ta;
                tmin_n = n;
            }
            if (tb < tmax)
            {
                tmax = tb;
                tmax_n = n;
            }
        }
    }

    if (tmax >= tmin && tmax > 0)
    {
        outside = true;
        if (tmin <= 0)
        {
            tmin = tmax;
            tmin_n = tmax_n;
            outside = false;
        }
        intersectionPoint = multiplyMV(box.transform, glm::vec4(getPointOnRay(q, tmin), 1.0f));
        normal = glm::normalize(multiplyMV(box.invTranspose, glm::vec4(tmin_n, 0.0f)));
        return glm::length(r.origin - intersectionPoint);
    }

    return -1;
}

__host__ __device__ float sphereIntersectionTest(
    Geom sphere,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    float radius = .5;

    glm::vec3 ro = multiplyMV(sphere.inverseTransform, glm::vec4(r.origin, 1.0f));
    glm::vec3 rd = glm::normalize(multiplyMV(sphere.inverseTransform, glm::vec4(r.direction, 0.0f)));

    Ray rt;
    rt.origin = ro;
    rt.direction = rd;

    float vDotDirection = glm::dot(rt.origin, rt.direction);
    float radicand = vDotDirection * vDotDirection - (glm::dot(rt.origin, rt.origin) - powf(radius, 2));
    if (radicand < 0)
    {
        return -1;
    }

    float squareRoot = sqrt(radicand);
    float firstTerm = -vDotDirection;
    float t1 = firstTerm + squareRoot;
    float t2 = firstTerm - squareRoot;

    float t = 0;
    if (t1 < 0 && t2 < 0)
    {
        return -1;
    }
    else if (t1 > 0 && t2 > 0)
    {
        t = min(t1, t2);
        outside = true;
    }
    else
    {
        t = max(t1, t2);
        outside = false;
    }

    glm::vec3 objspaceIntersection = getPointOnRay(rt, t);

    intersectionPoint = multiplyMV(sphere.transform, glm::vec4(objspaceIntersection, 1.f));
    normal = glm::normalize(multiplyMV(sphere.invTranspose, glm::vec4(objspaceIntersection, 0.f)));
    if (!outside)
    {
        normal = -normal;
    }

    return glm::length(r.origin - intersectionPoint);
}

__device__ bool intersectAABB(const Ray& ray, const glm::vec3& aabbMin, const glm::vec3& aabbMax) {
    glm::vec3 invDir = 1.0f / ray.direction;
    glm::vec3 t0 = (aabbMin - ray.origin) * invDir;
    glm::vec3 t1 = (aabbMax - ray.origin) * invDir;

    glm::vec3 tmin = glm::min(t0, t1);
    glm::vec3 tmax = glm::max(t0, t1);

    float tNear = glm::max(glm::max(tmin.x, tmin.y), tmin.z);
    float tFar = glm::min(glm::min(tmax.x, tmax.y), tmax.z);

    return tNear <= tFar && tFar > 0.0f;
}

__device__ float singleTriangleIntersectionTest(
    const Geom& geom,
    const TriangleMeshData& mesh,
    int triangleIndex,
    const Ray& r,
    glm::vec3& intersectionPoint,
    glm::vec3& normal,
    bool& outside)
{
    // Transform ray to object space
    glm::vec3 ro = glm::vec3(geom.inverseTransform * glm::vec4(r.origin, 1.0f));
    glm::vec3 rd = glm::normalize(glm::vec3(geom.inverseTransform * glm::vec4(r.direction, 0.0f)));

    const float EPS = 1e-7f;

    const unsigned int i0 = mesh.indices[triangleIndex * 3 + 0];
    const unsigned int i1 = mesh.indices[triangleIndex * 3 + 1];
    const unsigned int i2 = mesh.indices[triangleIndex * 3 + 2];

    const glm::vec3 v0(mesh.vertices[i0 * 3], mesh.vertices[i0 * 3 + 1], mesh.vertices[i0 * 3 + 2]);
    const glm::vec3 v1(mesh.vertices[i1 * 3], mesh.vertices[i1 * 3 + 1], mesh.vertices[i1 * 3 + 2]);
    const glm::vec3 v2(mesh.vertices[i2 * 3], mesh.vertices[i2 * 3 + 1], mesh.vertices[i2 * 3 + 2]);

    // Moller-Trumbore
    const glm::vec3 e1 = v1 - v0;
    const glm::vec3 e2 = v2 - v0;
    const glm::vec3 pvec = glm::cross(rd, e2);
    const float det = glm::dot(e1, pvec);

    if (fabsf(det) < EPS) return -1.0f;

    const float invDet = 1.0f / det;
    const glm::vec3 tvec = ro - v0;
    const float u = glm::dot(tvec, pvec) * invDet;

    if (u < 0.0f || u > 1.0f) return -1.0f;

    const glm::vec3 qvec = glm::cross(tvec, e1);
    const float v = glm::dot(rd, qvec) * invDet;

    if (v < 0.0f || (u + v) > 1.0f) return -1.0f;

    const float t = glm::dot(e2, qvec) * invDet;

    if (t <= EPS) return -1.0f;

    // Interpolate normals
    glm::vec3 objNormal;
    if (mesh.normals) {
        const glm::vec3 n0(mesh.normals[i0 * 3], mesh.normals[i0 * 3 + 1], mesh.normals[i0 * 3 + 2]);
        const glm::vec3 n1(mesh.normals[i1 * 3], mesh.normals[i1 * 3 + 1], mesh.normals[i1 * 3 + 2]);
        const glm::vec3 n2(mesh.normals[i2 * 3], mesh.normals[i2 * 3 + 1], mesh.normals[i2 * 3 + 2]);
        const float w = 1.0f - u - v;
        objNormal = glm::normalize(w * n0 + u * n1 + v * n2);
    }
    else {
        objNormal = glm::normalize(glm::cross(e1, e2));
    }

    // Transform back to world space
    const glm::vec3 Pobj = ro + rd * t;
    intersectionPoint = glm::vec3(geom.transform * glm::vec4(Pobj, 1.0f));
    normal = glm::normalize(glm::vec3(geom.invTranspose * glm::vec4(objNormal, 0.0f)));

    outside = glm::dot(r.direction, normal) < 0.0f;
    if (!outside) normal = -normal;

    return glm::length(intersectionPoint - r.origin);
}

__device__ float meshIntersectionTest(
    const Geom& geom,
    const TriangleMeshData& mesh,
    Ray r,
    glm::vec3& intersectionPoint,
    glm::vec3& normal,
    bool& outside)
{
    // Ray to object space
    const glm::vec3 ro = glm::vec3(geom.inverseTransform * glm::vec4(r.origin, 1.0f));
    const glm::vec3 rd = glm::normalize(glm::vec3(geom.inverseTransform * glm::vec4(r.direction, 0.0f)));

    const float EPS = 1e-7f;
    float tClosest = 1e30f;
    glm::vec3 bestNormal(0.0f);
    bool hit = false;

    for (int i = 0; i < mesh.triangleCount; ++i) {
        const unsigned int i0 = mesh.indices[i * 3 + 0];
        const unsigned int i1 = mesh.indices[i * 3 + 1];
        const unsigned int i2 = mesh.indices[i * 3 + 2];

        const glm::vec3 v0(mesh.vertices[i0 * 3 + 0], mesh.vertices[i0 * 3 + 1], mesh.vertices[i0 * 3 + 2]);
        const glm::vec3 v1(mesh.vertices[i1 * 3 + 0], mesh.vertices[i1 * 3 + 1], mesh.vertices[i1 * 3 + 2]);
        const glm::vec3 v2(mesh.vertices[i2 * 3 + 0], mesh.vertices[i2 * 3 + 1], mesh.vertices[i2 * 3 + 2]);

        const glm::vec3 e1 = v1 - v0;
        const glm::vec3 e2 = v2 - v0;

        // Moller-Trumbore
        const glm::vec3 pvec = glm::cross(rd, e2);
        const float det = glm::dot(e1, pvec);
        if (fabsf(det) < EPS) continue;
        const float invDet = 1.0f / det;

        const glm::vec3 tvec = ro - v0;
        const float u = glm::dot(tvec, pvec) * invDet;
        if (u < 0.0f || u > 1.0f) continue;

        const glm::vec3 qvec = glm::cross(tvec, e1);
        const float v = glm::dot(rd, qvec) * invDet;
        if (v < 0.0f || (u + v) > 1.0f) continue;

        const float t = glm::dot(e2, qvec) * invDet;
        if (t <= EPS) continue;

        if (t < tClosest) {
            tClosest = t;
            hit = true;

            // Interpolate normals if available; otherwise face normal
            if (mesh.normals) {
                const glm::vec3 n0(mesh.normals[i0 * 3 + 0], mesh.normals[i0 * 3 + 1], mesh.normals[i0 * 3 + 2]);
                const glm::vec3 n1(mesh.normals[i1 * 3 + 0], mesh.normals[i1 * 3 + 1], mesh.normals[i1 * 3 + 2]);
                const glm::vec3 n2(mesh.normals[i2 * 3 + 0], mesh.normals[i2 * 3 + 1], mesh.normals[i2 * 3 + 2]);
                const float w = 1.0f - u - v;
                bestNormal = glm::normalize(w * n0 + u * n1 + v * n2);
            } else {
                bestNormal = glm::normalize(glm::cross(e1, e2));
            }
        }
    }

    if (!hit) return -1.0f;

    // Object to world
    const glm::vec3 Pobj = ro + rd * tClosest;
    intersectionPoint = glm::vec3(geom.transform * glm::vec4(Pobj, 1.0f));
    normal = glm::normalize(glm::vec3(geom.invTranspose * glm::vec4(bestNormal, 0.0f)));

    // Orient and set outside
    outside = glm::dot(r.direction, normal) < 0.0f;
    if (!outside) normal = -normal;

    // Return world-space distance like your sphere/box tests
    return glm::length(intersectionPoint - r.origin);
}
