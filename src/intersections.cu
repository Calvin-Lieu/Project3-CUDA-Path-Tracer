#include "hip/hip_runtime.h"
#include "intersections.h"
#define GLM_ENABLE_EXPERIMENTAL
#include <glm/gtx/intersect.hpp>

__host__ __device__ float boxIntersectionTest(
    Geom box,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    Ray q;
    q.origin    =                multiplyMV(box.inverseTransform, glm::vec4(r.origin   , 1.0f));
    q.direction = glm::normalize(multiplyMV(box.inverseTransform, glm::vec4(r.direction, 0.0f)));

    float tmin = -1e38f;
    float tmax = 1e38f;
    glm::vec3 tmin_n;
    glm::vec3 tmax_n;
    for (int xyz = 0; xyz < 3; ++xyz)
    {
        float qdxyz = q.direction[xyz];
        /*if (glm::abs(qdxyz) > 0.00001f)*/
        {
            float t1 = (-0.5f - q.origin[xyz]) / qdxyz;
            float t2 = (+0.5f - q.origin[xyz]) / qdxyz;
            float ta = glm::min(t1, t2);
            float tb = glm::max(t1, t2);
            glm::vec3 n;
            n[xyz] = t2 < t1 ? +1 : -1;
            if (ta > 0 && ta > tmin)
            {
                tmin = ta;
                tmin_n = n;
            }
            if (tb < tmax)
            {
                tmax = tb;
                tmax_n = n;
            }
        }
    }

    if (tmax >= tmin && tmax > 0)
    {
        outside = true;
        if (tmin <= 0)
        {
            tmin = tmax;
            tmin_n = tmax_n;
            outside = false;
        }
        intersectionPoint = multiplyMV(box.transform, glm::vec4(getPointOnRay(q, tmin), 1.0f));
        normal = glm::normalize(multiplyMV(box.invTranspose, glm::vec4(tmin_n, 0.0f)));
        return glm::length(r.origin - intersectionPoint);
    }

    return -1;
}

__host__ __device__ float sphereIntersectionTest(
    Geom sphere,
    Ray r,
    glm::vec3& intersectionPoint,
    glm::vec3& normal,
    bool& outside)
{
    // Define the sphere radius
    constexpr float radius = 0.5f;

    // Transform ray into object space
    glm::vec3 ro = multiplyMV(sphere.inverseTransform, glm::vec4(r.origin, 1.0f));
    glm::vec3 rd = glm::normalize(multiplyMV(sphere.inverseTransform, glm::vec4(r.direction, 0.0f)));

    // Quadratic coefficients for the sphere intersection
    float a = glm::dot(rd, rd);
    float b = 2.0f * glm::dot(ro, rd);
    float c = glm::dot(ro, ro) - radius * radius;

    // Compute discriminant to check intersection
    float discriminant = b * b - 4.0f * a * c;
    if (discriminant < 0.0f) {
        // No real roots; the ray misses the sphere
        return -1.0f;
    }

    // Compute intersection points using the quadratic formula
    float sqrtDiscriminant = sqrtf(discriminant);
    float t1 = (-b - sqrtDiscriminant) / (2.0f * a);
    float t2 = (-b + sqrtDiscriminant) / (2.0f * a);

    // Find the nearest valid intersection point
    float t = (t1 > 0.0f) ? t1 : (t2 > 0.0f ? t2 : -1.0f);
    if (t < 0.0f) {
        // Both intersections are behind the ray origin
        return -1.0f;
    }

    // Set outside flag: if we used t1, we're outside; if t2, we're inside
    outside = (t1 > 0.0f);

    // Compute object-space intersection point
    glm::vec3 objSpaceIntersection = ro + t * rd;

    // Transform intersection point and normal back to world space
    intersectionPoint = multiplyMV(sphere.transform, glm::vec4(objSpaceIntersection, 1.0f));
    normal = glm::normalize(multiplyMV(sphere.invTranspose, glm::vec4(objSpaceIntersection, 0.0f)));

    // Flip normal if hitting from inside
    if (!outside) {
        normal = -normal;
    }

    // Return the distance from the ray origin to the intersection point
    return glm::length(r.origin - intersectionPoint);
}

__device__ bool intersectAABB(const Ray& ray, const glm::vec3& aabbMin, const glm::vec3& aabbMax) {
    glm::vec3 invDir = 1.0f / ray.direction;
    glm::vec3 t0 = (aabbMin - ray.origin) * invDir;
    glm::vec3 t1 = (aabbMax - ray.origin) * invDir;

    glm::vec3 tmin = glm::min(t0, t1);
    glm::vec3 tmax = glm::max(t0, t1);

    float tNear = glm::max(glm::max(tmin.x, tmin.y), tmin.z);
    float tFar = glm::min(glm::min(tmax.x, tmax.y), tmax.z);

    return tNear <= tFar && tFar > 0.0f;
}

__device__ float singleTriangleIntersectionTest(
    const Geom& geom,
    const TriangleMeshData& mesh,
    int triangleIndex,
    const Ray& r,
    glm::vec3& intersectionPoint,
    glm::vec3& normal,
    bool& outside)
{
    // Transform ray to object space
    glm::vec3 ro = glm::vec3(geom.inverseTransform * glm::vec4(r.origin, 1.0f));
    glm::vec3 rd = glm::normalize(glm::vec3(geom.inverseTransform * glm::vec4(r.direction, 0.0f)));

    const float EPS = 1e-7f;

    const unsigned int i0 = mesh.indices[triangleIndex * 3 + 0];
    const unsigned int i1 = mesh.indices[triangleIndex * 3 + 1];
    const unsigned int i2 = mesh.indices[triangleIndex * 3 + 2];

    const glm::vec3 v0(mesh.vertices[i0 * 3], mesh.vertices[i0 * 3 + 1], mesh.vertices[i0 * 3 + 2]);
    const glm::vec3 v1(mesh.vertices[i1 * 3], mesh.vertices[i1 * 3 + 1], mesh.vertices[i1 * 3 + 2]);
    const glm::vec3 v2(mesh.vertices[i2 * 3], mesh.vertices[i2 * 3 + 1], mesh.vertices[i2 * 3 + 2]);

    // Moller-Trumbore
    const glm::vec3 e1 = v1 - v0;
    const glm::vec3 e2 = v2 - v0;
    const glm::vec3 pvec = glm::cross(rd, e2);
    const float det = glm::dot(e1, pvec);

    if (fabsf(det) < EPS) return -1.0f;

    const float invDet = 1.0f / det;
    const glm::vec3 tvec = ro - v0;
    const float u = glm::dot(tvec, pvec) * invDet;

    if (u < 0.0f || u > 1.0f) return -1.0f;

    const glm::vec3 qvec = glm::cross(tvec, e1);
    const float v = glm::dot(rd, qvec) * invDet;

    if (v < 0.0f || (u + v) > 1.0f) return -1.0f;

    const float t = glm::dot(e2, qvec) * invDet;

    if (t <= EPS) return -1.0f;

    // Interpolate normals
    glm::vec3 objNormal;
    if (mesh.normals) {
        const glm::vec3 n0(mesh.normals[i0 * 3], mesh.normals[i0 * 3 + 1], mesh.normals[i0 * 3 + 2]);
        const glm::vec3 n1(mesh.normals[i1 * 3], mesh.normals[i1 * 3 + 1], mesh.normals[i1 * 3 + 2]);
        const glm::vec3 n2(mesh.normals[i2 * 3], mesh.normals[i2 * 3 + 1], mesh.normals[i2 * 3 + 2]);
        const float w = 1.0f - u - v;
        objNormal = glm::normalize(w * n0 + u * n1 + v * n2);
    }
    else {
        objNormal = glm::normalize(glm::cross(e1, e2));
    }

    // Transform back to world space
    const glm::vec3 Pobj = ro + rd * t;
    intersectionPoint = glm::vec3(geom.transform * glm::vec4(Pobj, 1.0f));
    normal = glm::normalize(glm::vec3(geom.invTranspose * glm::vec4(objNormal, 0.0f)));

    outside = glm::dot(r.direction, normal) < 0.0f;
    if (!outside) normal = -normal;

    return glm::length(intersectionPoint - r.origin);
}

__device__ float meshIntersectionTest(
    const Geom& geom,
    const TriangleMeshData& mesh,
    Ray r,
    glm::vec3& intersectionPoint,
    glm::vec3& normal,
    bool& outside)
{
    // Ray to object space
    const glm::vec3 ro = glm::vec3(geom.inverseTransform * glm::vec4(r.origin, 1.0f));
    const glm::vec3 rd = glm::normalize(glm::vec3(geom.inverseTransform * glm::vec4(r.direction, 0.0f)));

    const float EPS = 1e-7f;
    float tClosest = 1e30f;
    glm::vec3 bestNormal(0.0f);
    bool hit = false;

    for (int i = 0; i < mesh.triangleCount; ++i) {
        const unsigned int i0 = mesh.indices[i * 3 + 0];
        const unsigned int i1 = mesh.indices[i * 3 + 1];
        const unsigned int i2 = mesh.indices[i * 3 + 2];

        const glm::vec3 v0(mesh.vertices[i0 * 3 + 0], mesh.vertices[i0 * 3 + 1], mesh.vertices[i0 * 3 + 2]);
        const glm::vec3 v1(mesh.vertices[i1 * 3 + 0], mesh.vertices[i1 * 3 + 1], mesh.vertices[i1 * 3 + 2]);
        const glm::vec3 v2(mesh.vertices[i2 * 3 + 0], mesh.vertices[i2 * 3 + 1], mesh.vertices[i2 * 3 + 2]);

        const glm::vec3 e1 = v1 - v0;
        const glm::vec3 e2 = v2 - v0;

        // Moller-Trumbore
        const glm::vec3 pvec = glm::cross(rd, e2);
        const float det = glm::dot(e1, pvec);
        if (fabsf(det) < EPS) continue;
        const float invDet = 1.0f / det;

        const glm::vec3 tvec = ro - v0;
        const float u = glm::dot(tvec, pvec) * invDet;
        if (u < 0.0f || u > 1.0f) continue;

        const glm::vec3 qvec = glm::cross(tvec, e1);
        const float v = glm::dot(rd, qvec) * invDet;
        if (v < 0.0f || (u + v) > 1.0f) continue;

        const float t = glm::dot(e2, qvec) * invDet;
        if (t <= EPS) continue;

        if (t < tClosest) {
            tClosest = t;
            hit = true;

            // Interpolate normals if available; otherwise face normal
            if (mesh.normals) {
                const glm::vec3 n0(mesh.normals[i0 * 3 + 0], mesh.normals[i0 * 3 + 1], mesh.normals[i0 * 3 + 2]);
                const glm::vec3 n1(mesh.normals[i1 * 3 + 0], mesh.normals[i1 * 3 + 1], mesh.normals[i1 * 3 + 2]);
                const glm::vec3 n2(mesh.normals[i2 * 3 + 0], mesh.normals[i2 * 3 + 1], mesh.normals[i2 * 3 + 2]);
                const float w = 1.0f - u - v;
                bestNormal = glm::normalize(w * n0 + u * n1 + v * n2);
            } else {
                bestNormal = glm::normalize(glm::cross(e1, e2));
            }
        }
    }

    if (!hit) return -1.0f;

    // Object to world
    const glm::vec3 Pobj = ro + rd * tClosest;
    intersectionPoint = glm::vec3(geom.transform * glm::vec4(Pobj, 1.0f));
    normal = glm::normalize(glm::vec3(geom.invTranspose * glm::vec4(bestNormal, 0.0f)));

    // Orient and set outside
    outside = glm::dot(r.direction, normal) < 0.0f;
    if (!outside) normal = -normal;

    // Return world-space distance like your sphere/box tests
    return glm::length(intersectionPoint - r.origin);
}
