#include "hip/hip_runtime.h"
#include "pathHelpers.h"
#include <thrust/random.h>
#include "intersections.h"


__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

__device__ float rr_luminance(const glm::vec3& c) {
    return 0.2126f * c.x + 0.7152f * c.y + 0.0722f * c.z;
}

__device__ void applyRussianRoulette(PathSegment& ps,
    int depth,
    int rrStartDepth,
    thrust::default_random_engine& rng)
{
    if (depth < rrStartDepth) return;

    thrust::uniform_real_distribution<float> u01(0.f, 1.f);

    // Termination probability (increases as throughput dims)
    float q = fmaxf(0.05f, 1.0f - rr_luminance(ps.color));

    if (u01(rng) < q) {
        ps.color = glm::vec3(0.0f);
        ps.remainingBounces = 0;
    }
    else {
        ps.color /= (1.0f - q);
    }
}
