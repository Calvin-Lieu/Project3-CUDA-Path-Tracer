#include "hip/hip_runtime.h"
#include "interactions.h"

#include "utilities.h"

#include <thrust/random.h>
#include <thrust/swap.h>
#include <hip/hip_math_constants.h>
#include <glm/gtx/norm.hpp>
#include <glm/gtx/component_wise.hpp>



__host__ __device__ glm::vec3 calculateRandomDirectionInHemisphere(
    glm::vec3 normal,
    thrust::default_random_engine& rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);

    float up = sqrt(u01(rng)); // cos(theta)
    float over = sqrt(1 - up * up); // sin(theta)
    float around = u01(rng) * TWO_PI;

    // Find a direction that is not the normal based off of whether or not the
    // normal's components are all equal to sqrt(1/3) or whether or not at
    // least one component is less than sqrt(1/3). Learned this trick from
    // Peter Kutz.

    glm::vec3 directionNotNormal;
    if (abs(normal.x) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(1, 0, 0);
    }
    else if (abs(normal.y) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(0, 1, 0);
    }
    else
    {
        directionNotNormal = glm::vec3(0, 0, 1);
    }

    // Use not-normal direction to generate two perpendicular directions
    glm::vec3 perpendicularDirection1 =
        glm::normalize(glm::cross(normal, directionNotNormal));
    glm::vec3 perpendicularDirection2 =
        glm::normalize(glm::cross(normal, perpendicularDirection1));

    return up * normal
        + cos(around) * over * perpendicularDirection1
        + sin(around) * over * perpendicularDirection2;
}


// ------------------------------------------------------------
// BRDFs
// ------------------------------------------------------------

// Diffuse (Lambertian)
__host__ __device__ void diffuseBRDF(
    PathSegment& pathSegment,
    glm::vec3 normal,
    const Material& m,
    thrust::default_random_engine& rng,
    glm::vec3 intersect)
{
    glm::vec3 wi = calculateRandomDirectionInHemisphere(normal, rng);

    float cosTheta = fmaxf(0.f, glm::dot(wi, normal));
    float pdf = (cosTheta > 0.f) ? (cosTheta / HIP_PI_F) : 1e-6f;

    pathSegment.color *= m.color;
    pathSegment.prevBsdfPdf = pdf;

    pathSegment.ray.origin = intersect + normal * 1e-3f;
    pathSegment.ray.direction = wi;


}

// GGX Specular / Metallic
__device__ void ggxSpecularBRDF(
    PathSegment& pathSegment,
    glm::vec3 normal,
    const Material& m,
    thrust::default_random_engine& rng,
    glm::vec3 intersect)
{
    float eps = 1e-3f;
    glm::vec3 n = glm::normalize(normal);
    glm::vec3 wo = -glm::normalize(pathSegment.ray.direction); 

    float metallic = fminf(fmaxf(m.metallic, 0.f), 1.f);
    glm::vec3 F0 = glm::mix(glm::vec3(0.04f), m.color, metallic);

    float rough = fminf(fmaxf(m.roughness, 0.f), 1.f);
    float alpha = fmaxf(rough * rough, 1e-3f);

    // Ideal mirror branch
    if (alpha < 1e-4f) {
        glm::vec3 wi = glm::reflect(-wo, n);
        float NoV = fmaxf(1e-6f, fabsf(glm::dot(n, wo)));
        glm::vec3 F = Fresnel_Schlick(NoV, F0);

        pathSegment.ray.origin = intersect + n * eps;
        pathSegment.ray.direction = glm::normalize(wi);

        pathSegment.prevBsdfPdf = 0.0f;
        pathSegment.prevWasDelta = 1;
        pathSegment.color *= F;
        return;
    }

    // Rough GGX
    glm::vec3 h = sampleGGX_H(n, alpha, rng);
    glm::vec3 wi = glm::reflect(-wo, h);

    float NoV = fmaxf(1e-6f, fabsf(glm::dot(n, wo)));
    float NoL = fmaxf(0.0f, glm::dot(n, wi));
    if (NoL <= 0.0f) {
        pathSegment.color *= m.color;
        pathSegment.ray.direction = calculateRandomDirectionInHemisphere(n, rng);
        pathSegment.ray.origin = intersect + n * eps;
        pathSegment.prevBsdfPdf = HIP_PI_F;
        return;
    }
    float NoH = fmaxf(1e-6f, fabsf(glm::dot(n, h)));
    float HoV = fmaxf(1e-6f, fabsf(glm::dot(h, wo)));

    float D = D_GGX(NoH, alpha);
    float G = G_SmithGGX(NoV, NoL, alpha);
    glm::vec3 F = Fresnel_Schlick(HoV, F0);

    glm::vec3 f = (D * G) * F / (4.0f * NoV * NoL + 1e-6f);

    float pdf = (D * NoH) / (4.0f * HoV + 1e-6f);
    pdf = fmaxf(pdf, 1e-6f);

    pathSegment.color *= F * G * HoV / (NoV * NoH);
    pathSegment.prevBsdfPdf = pdf;

    pathSegment.ray.origin = intersect + n * eps;
    pathSegment.ray.direction = glm::normalize(wi);
}

// Dielectric (Glass)
__host__ __device__ void dielectricBRDF(
    PathSegment& pathSegment,
    glm::vec3 normal,
    const Material& m,
    thrust::default_random_engine& rng,
    glm::vec3 intersect)
{
    glm::vec3 incidentDir = glm::normalize(pathSegment.ray.direction);

    // Check if ray is entering or exiting
    bool isInside = glm::dot(incidentDir, normal) > 0.0f;
    float iorFrom = isInside ? m.indexOfRefraction : 1.0f;   // medium we are coming from
    float iorTo = isInside ? 1.0f : m.indexOfRefraction;   // medium we are going into
    float eta = iorFrom / iorTo;                                // relative index of refraction

    glm::vec3 orientedNormal = isInside ? -normal : normal;

    float cosIncident = glm::clamp(-glm::dot(incidentDir, orientedNormal), 0.f, 1.f);

    // Reflection and refraction directions
    glm::vec3 reflectDir = glm::reflect(incidentDir, orientedNormal);
    glm::vec3 refractDir = glm::refract(incidentDir, orientedNormal, eta);

    // Fresnel reflectance (probability of reflection)
    float reflectance = Fresnel_Schlick(cosIncident, iorFrom, iorTo);

    thrust::uniform_real_distribution<float> u01(0, 1);
    float xi = u01(rng);

    if (xi < reflectance || glm::length2(refractDir) < 1e-10f) {
        pathSegment.ray.direction = glm::normalize(reflectDir);
        pathSegment.ray.origin = intersect + orientedNormal * 1e-3f;
    }
    else {
        pathSegment.ray.direction = glm::normalize(refractDir);
        pathSegment.ray.origin = intersect - orientedNormal * 1e-3f;
        pathSegment.color *= m.color;
    }

    pathSegment.prevBsdfPdf = 0.0f;
    pathSegment.prevWasDelta = 0;
}

// ------------------------------------------------------------
// Scatter Dispatcher
// ------------------------------------------------------------
__device__ void scatterRay(
    PathSegment& pathSegment,
    glm::vec3 P,
    glm::vec3 N,
    const Material& m,
    thrust::default_random_engine& rng,
    int depth,
    int materialId)
{
    // Dielectric path 
    if (m.hasRefractive > 0.5f || m.transmission > 0.0f) {
        dielectricBRDF(pathSegment, N, m, rng, P);
        //atomicAdd(&gRefractiveCounts[materialId], 1);
        pathSegment.remainingBounces--;
        return;
    }

    // Opaque metallic-roughness mixture 
    // Clamp inputs
    float metallic  = glm::clamp(m.metallic,  0.0f, 1.0f);
    float roughness = glm::clamp(m.roughness, 0.04f, 1.0f);

    // Base reflectance:
    glm::vec3 F0 = glm::mix(glm::vec3(0.04f), m.color, metallic);

    // Lobe weights:
    //   Metals: no diffuse 
    float Favg = 0.3333f * (F0.x + F0.y + F0.z);  // average Fresnel at normal incidence
    float diffuseWeight = (1.0f - metallic) * (1.0f - Favg);
    diffuseWeight = glm::clamp(diffuseWeight, 0.0f, 1.0f);
    float specWeight = 1.0f - diffuseWeight;

    // Sample which lobe to use
    thrust::uniform_real_distribution<float> u01(0,1);
    float xi = u01(rng);

    if (xi < specWeight) {
        // GGX specular
        ggxSpecularBRDF(pathSegment, N, m, rng, P);
        //atomicAdd(&gSpecularCounts[materialId], 1);
    } else {
        // Lambertian diffuse
        diffuseBRDF(pathSegment, N, m, rng, P);
        //atomicAdd(&gDiffuseCounts[materialId], 1);
    }

    pathSegment.remainingBounces--;
}
