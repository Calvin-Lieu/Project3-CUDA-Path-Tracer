#include "hip/hip_runtime.h"
#include "interactions.h"

#include "utilities.h"

#include <thrust/random.h>
#include <thrust/swap.h>
#include <hip/hip_math_constants.h>
#include <glm/gtx/norm.hpp>


// ----- Microfacet GGX helpers (reflection) -----
__device__ inline void makeONB(const glm::vec3& n, glm::vec3& t, glm::vec3& b) {
    if (fabsf(n.z) < 0.999f) {
        t = glm::normalize(glm::vec3(-n.y, n.x, 0.0f));
    }
    else {
        t = glm::normalize(glm::vec3(0.0f, 1.0f, 0.0f));
    }
    b = glm::cross(n, t);
}

__device__ inline float D_GGX(float NoH, float alpha) {
    // Trowbridge-Reitz GGX NDF
    float a2 = alpha * alpha;
    float d = NoH * NoH * (a2 - 1.0f) + 1.0f;
    return a2 / (HIP_PI_F * d * d);
}

__device__ inline float Lambda_GGX(float cosTheta, float alpha) {
    // Smith's lambda term for GGX
    float a = alpha;
    float cos2 = cosTheta * cosTheta;
    float sin2 = fmaxf(0.0f, 1.0f - cos2);
    float tan2 = (cos2 > 0.0f) ? (sin2 / cos2) : 1e20f;
    return (-1.0f + sqrtf(1.0f + a * a * tan2)) * 0.5f;
}

__device__ inline float G_SmithGGX(float NoV, float NoL, float alpha) {
    float a2 = alpha * alpha;
    float GGXV = NoV * sqrtf(a2 + (1.0f - a2) * NoL * NoL);
    float GGXL = NoL * sqrtf(a2 + (1.0f - a2) * NoV * NoV);
    return 2.0f * NoL * NoV / (GGXV + GGXL + 1e-6f);
}

// --- Vec3 version (For metals/rough GGX) ---
__host__ __device__ inline glm::vec3 Fresnel_Schlick(
    float cosTheta,
    const glm::vec3& F0)
{
    float m = fminf(fmaxf(1.0f - cosTheta, 0.0f), 1.0f);
    float m2 = m * m;
    float m5 = m2 * m2 * m;
    return F0 + (glm::vec3(1.0f) - F0) * m5;
}

// --- Float version (Ffor glass/dielectrics) ---
__host__ __device__ inline float Fresnel_Schlick(
    float cosTheta,
    float etaI,
    float etaT)
{
    float r0 = (etaI - etaT) / (etaI + etaT);
    r0 = r0 * r0;
    return r0 + (1.0f - r0) * powf(1.0f - cosTheta, 5.0f);
}

// Sample GGX half-vector (basic Trowbridge-Reitz)
__device__ inline glm::vec3 sampleGGX_H(const glm::vec3& n, float alpha,
    thrust::default_random_engine& rng) {
    thrust::uniform_real_distribution<float> u01(0.f, 1.f);
    float u1 = u01(rng);
    float u2 = u01(rng);

    float phi = 2.0f * HIP_PI_F * u1;
    float a2 = alpha * alpha;
    float cosTheta = sqrtf((1.0f - u2) / (1.0f + (a2 - 1.0f) * u2));
    float sinTheta = sqrtf(fmaxf(0.0f, 1.0f - cosTheta * cosTheta));

    glm::vec3 t, b;
    makeONB(n, t, b);
    // local -> world
    return glm::normalize(
        sinTheta * cosf(phi) * t +
        sinTheta * sinf(phi) * b +
        cosTheta * n);
}


__host__ __device__ glm::vec3 calculateRandomDirectionInHemisphere(
    glm::vec3 normal,
    thrust::default_random_engine &rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);

    float up = sqrt(u01(rng)); // cos(theta)
    float over = sqrt(1 - up * up); // sin(theta)
    float around = u01(rng) * TWO_PI;

    // Find a direction that is not the normal based off of whether or not the
    // normal's components are all equal to sqrt(1/3) or whether or not at
    // least one component is less than sqrt(1/3). Learned this trick from
    // Peter Kutz.

    glm::vec3 directionNotNormal;
    if (abs(normal.x) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(1, 0, 0);
    }
    else if (abs(normal.y) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(0, 1, 0);
    }
    else
    {
        directionNotNormal = glm::vec3(0, 0, 1);
    }

    // Use not-normal direction to generate two perpendicular directions
    glm::vec3 perpendicularDirection1 =
        glm::normalize(glm::cross(normal, directionNotNormal));
    glm::vec3 perpendicularDirection2 =
        glm::normalize(glm::cross(normal, perpendicularDirection1));

    return up * normal
        + cos(around) * over * perpendicularDirection1
        + sin(around) * over * perpendicularDirection2;
}


// ------------------------------------------------------------
// BRDFs
// ------------------------------------------------------------

// Diffuse (Lambertian)
__host__ __device__ void diffuseBRDF(
    PathSegment& pathSegment,
    glm::vec3 normal,
    const Material& m,
    thrust::default_random_engine& rng,
    glm::vec3 intersect)
{
    glm::vec3 wi = calculateRandomDirectionInHemisphere(normal, rng);

    float cosTheta = fmaxf(0.f, glm::dot(wi, normal));
    float pdf = (cosTheta > 0.f) ? (cosTheta / HIP_PI_F) : 1e-6f;

    pathSegment.color *= m.color;
    pathSegment.prevBsdfPdf = pdf;

    pathSegment.ray.origin = intersect + normal * 1e-3f;
    pathSegment.ray.direction = wi;

    
}

// GGX Specular / Metallic
__device__ void ggxSpecularBRDF(
    PathSegment& pathSegment,
    glm::vec3 normal,
    const Material& m,
    thrust::default_random_engine& rng,
    glm::vec3 intersect)
{
    float eps = 1e-3f;
    glm::vec3 n = glm::normalize(normal);
    glm::vec3 wo = -glm::normalize(pathSegment.ray.direction); // outgoing/view

    float metallic = fminf(fmaxf(m.metallic, 0.f), 1.f);
    glm::vec3 F0 = glm::mix(glm::vec3(0.04f), m.color, metallic);

    float rough = fminf(fmaxf(m.roughness, 0.f), 1.f);
    float alpha = fmaxf(rough * rough, 1e-3f);

    // Ideal mirror branch
    if (alpha < 1e-4f) {
        glm::vec3 wi = glm::reflect(-wo, n);
        float NoV = fmaxf(1e-6f, fabsf(glm::dot(n, wo)));
        glm::vec3 F = Fresnel_Schlick(NoV, F0);

        pathSegment.ray.origin = intersect + n * eps;
        pathSegment.ray.direction = glm::normalize(wi);

        pathSegment.prevBsdfPdf = 0.0f;
        pathSegment.prevWasDelta = 1;
        pathSegment.color *= F;
        return;
    }

    // Rough GGX
    glm::vec3 h = sampleGGX_H(n, alpha, rng);
    glm::vec3 wi = glm::reflect(-wo, h);

    float NoV = fmaxf(1e-6f, fabsf(glm::dot(n, wo)));
    float NoL = fmaxf(0.0f, glm::dot(n, wi));
    if (NoL <= 0.0f) {
        pathSegment.color *= m.color;
        pathSegment.ray.direction = calculateRandomDirectionInHemisphere(n, rng);
        pathSegment.ray.origin = intersect + n * eps;
        pathSegment.prevBsdfPdf = HIP_PI_F;
        return;
    }
    float NoH = fmaxf(1e-6f, fabsf(glm::dot(n, h)));
    float HoV = fmaxf(1e-6f, fabsf(glm::dot(h, wo)));

    float D = D_GGX(NoH, alpha);
    float G = G_SmithGGX(NoV, NoL, alpha);
    glm::vec3 F = Fresnel_Schlick(HoV, F0);

    glm::vec3 f = (D * G) * F / (4.0f * NoV * NoL + 1e-6f);

    float pdf = (D * NoH) / (4.0f * HoV + 1e-6f);
    pdf = fmaxf(pdf, 1e-6f);

    pathSegment.color *= F * G * HoV / (NoV * NoH);
    pathSegment.prevBsdfPdf = pdf;

    pathSegment.ray.origin = intersect + n * eps;
    pathSegment.ray.direction = glm::normalize(wi);
}

// Dielectric (Glass)
__host__ __device__ void dielectricBRDF(
    PathSegment& pathSegment,
    glm::vec3 normal,
    const Material& m,
    thrust::default_random_engine& rng,
    glm::vec3 intersect)
{
    glm::vec3 incidentDir = glm::normalize(pathSegment.ray.direction);

    // Check if ray is entering or exiting
    bool isInside = glm::dot(incidentDir, normal) > 0.0f;
    float iorFrom = isInside ? m.indexOfRefraction : 1.0f;   // medium we are coming from
    float iorTo = isInside ? 1.0f : m.indexOfRefraction;   // medium we are going into
    float eta = iorFrom / iorTo;                                // relative index of refraction

    glm::vec3 orientedNormal = isInside ? -normal : normal;

    float cosIncident = glm::clamp(-glm::dot(incidentDir, orientedNormal), 0.f, 1.f);

    // Reflection and refraction directions
    glm::vec3 reflectDir = glm::reflect(incidentDir, orientedNormal);
    glm::vec3 refractDir = glm::refract(incidentDir, orientedNormal, eta);

    // Fresnel reflectance (probability of reflection)
    float reflectance = Fresnel_Schlick(cosIncident, iorFrom, iorTo);

    thrust::uniform_real_distribution<float> u01(0, 1);
    float xi = u01(rng);

    if (xi < reflectance  || glm::length2(refractDir) < 1e-10f) {
        pathSegment.ray.direction = glm::normalize(reflectDir);
        pathSegment.ray.origin = intersect + orientedNormal * 1e-3f;
    }
    else {
        pathSegment.ray.direction = glm::normalize(refractDir);
        pathSegment.ray.origin = intersect - orientedNormal * 1e-3f;
        pathSegment.color *= m.color;
    }

    pathSegment.prevBsdfPdf = 0.0f;
    pathSegment.prevWasDelta = 0;
}

// ------------------------------------------------------------
// Scatter Dispatcher
// ------------------------------------------------------------
__device__ void scatterRay(
    PathSegment& pathSegment,
    glm::vec3 intersect,
    glm::vec3 normal,
    const Material& m,
    thrust::default_random_engine& rng)
{
    if (m.hasReflective > 0.0f || m.metallic > 0.0f) {
        // GGX specular/metallic
		//printf("GGX Specular/Metallic\n");
        ggxSpecularBRDF(pathSegment, normal, m, rng, intersect);
    }
    else if (m.hasRefractive > 0.0f) {
        // Dielectric glass
		//printf("Dielectric Glass\n");
        dielectricBRDF(pathSegment, normal, m, rng, intersect);
    }
    else {
        // Diffuse
		//printf("Diffuse\n");
        diffuseBRDF(pathSegment, normal, m, rng, intersect);
    }

    pathSegment.remainingBounces--;
}
